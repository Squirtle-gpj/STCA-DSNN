#include "math_functions.h"
#include "hip/hip_runtime.h"


__global__ void compute_u_kernel( double *u, const double * input, const int T_size, const int Numcurlayer,  const double decay1, const double decay2)
{


	int afferent_id = blockIdx.x*blockDim.x +threadIdx.x;
	if(afferent_id > Numcurlayer - 1 ) {return;}
	int curid = afferent_id*T_size;
    int endid = curid + T_size;
    double m = 0;
    double s = 0;
    // double e = 0;
    double V = 0;

    // bool fired_pre = false;


    while(curid < endid)
    {
			m = m*decay1;
			s = s*decay2;
			//now, V is a tmp
			V = input[curid];
			if(V != 0)
			{
				m = m + V;
				s = s + V;
			}
			// e = e*decay1;
			// if (fired_pre)
			// {
			// 	e = e + threshold;
			// }
			// now, fired_pre is fired_cur.

			V = m -s;

			u[curid] = V;
			// fired_pre = (V > threshold);
			// if(fired_pre)
			// {
			// 	output[curid] = 1.0;
			// }
			curid++;

    }


}
