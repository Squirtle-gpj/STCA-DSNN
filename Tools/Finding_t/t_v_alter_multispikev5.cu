#include "math_functions.h"
#include "hip/hip_runtime.h"

//t+位于发放脉冲数最少的spike cluster的最后一个脉冲发放点
//t-位于不在spike_cluster中的阈�?下最高极大�?�?
//t+λ�ڷ������������ٵ�spike cluster�����һ�����巢�ŵ�
//t-λ�ڲ���spike_cluster�е���ֵ����߼���ֵ��
__global__ void t_v_alter_multispikev3( double *output, double *u, int *nSpikeClusters,int *t_alter, double *direction,const int *desired,const double * input, const int T_size, const int Numcurlayer,  const double decay1, const double decay2, const double threshold, const int interval)
{


	int neuron_id = blockIdx.x*blockDim.x +threadIdx.x;
	if(neuron_id > Numcurlayer - 1 ) {return;}
	int curid = neuron_id*T_size;
  int endid = curid + T_size;
  double m = 0;
  double s = 0;
  double e = 0;
  double V = 0;
	int nSpike_clusters = 0;
	double nSpikes = 0;
	double up_min = 1.79769313486231570E+308;
	double down_max = -1.79769313486231570E+308;
	double t_up_min = endid-1;
	double t_down_max = t_up_min;
	double V_nothr[2] = {0,0};// 0: 上一时刻的nothr_V - 上上�?��刻的nothr_V; 1: 上一时刻的nothr_V
	bool fired_pre = false;
	bool incluster = false;
	int dur = 0;


    while(curid < endid)
    {
			m = m*decay1;
			s = s*decay2;
			//now, V is a tmp
			V = input[curid];
			if(V != 0)
			{
				m = m + V;
				s = s + V;
			}
			e = e*decay1;
			if (fired_pre)
			{
				e = e + threshold;
			}
			// now, fired_pre is fired_cur.

			V = m -s - e;
			u[curid] = V;
			fired_pre = (V > threshold);
			if(fired_pre)
			{
				output[curid] = 1.0;
				incluster = true;
				dur = 0;
				nSpikes += V;
				 // nSpikes++;

			}else{
				dur++;
			}
			if(((dur>=interval)||(curid >=(endid-1)))&&(incluster == true)){
				nSpike_clusters++;
				incluster = false;
				if(nSpikes < up_min){
					up_min = nSpikes;
					t_up_min = curid-1-dur;
				}
				nSpikes = 0;

			}

			if((V_nothr[0]>0)&&(V_nothr[1] >V)){
				// output[curid] = 1.0;
				// if((V_nothr[1]>threshold)&&(V_nothr[1]<up_min)){
				// 	// output[curid] = 2.0;
				// 	up_min = V_nothr[1];
				// 	t_up_min = curid-1;
				// }
				// else
				if((V_nothr[1]<=threshold)&&(V_nothr[1]>down_max)&&(incluster == false)){
					// output[curid] = 3.0;
					down_max = V;
					t_down_max = curid-1;
				}

			}
			V_nothr[0] = V - V_nothr[1];
			V_nothr[1] = V;
			curid++;

    }

	nSpikeClusters[neuron_id] = nSpike_clusters;
	if(nSpike_clusters > desired[neuron_id]){
		direction[neuron_id] = -1;
		t_alter[neuron_id] = t_up_min -endid + T_size+1;//transform to matlab
	}
	else if(nSpike_clusters < desired[neuron_id]){
		direction[neuron_id] = 1;
		t_alter[neuron_id] = t_down_max -endid + T_size+1;
	}


}
